#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include "kernels.cuh"

void random_init(float *array, int M, int N) {
    for(int i = 0; i < M * N; i++)
        array[i] = (float)rand() / RAND_MAX;
}

int main() {
    srand(time(NULL));

    int M = 4096, N = 4096, K = 4096;

    float *h_matA, *h_matB, *h_matC, *h_matD;
    
    int matA_size = M * N * sizeof(float);
    int matB_size = N * K * sizeof(float);
    int matC_size = M * K * sizeof(float);
    int matD_size = M * K * sizeof(float);

    h_matA = (float*)malloc(matA_size);
    h_matB = (float*)malloc(matB_size);
    h_matC = (float*)malloc(matC_size);
    h_matD = (float*)malloc(matC_size);

    random_init(h_matA, M, N);
    random_init(h_matB, N, K);
    random_init(h_matC, M, K);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    float sec = 0.0;
    float *d_matA, *d_matB, *d_matC, *d_matD;

    hipEventRecord(start);
    hipMalloc((void**)&d_matA, matA_size);
    hipMalloc((void**)&d_matB, matB_size);
    hipMalloc((void**)&d_matC, matC_size);
    hipMalloc((void**)&d_matD, matC_size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&sec, start, stop);
    printf(">> GPU memory allocation time: %.3f\n", sec);

    hipEventRecord(start);
    hipMemcpy(d_matA, h_matA, matA_size, hipMemcpyHostToDevice);
    hipMemcpy(d_matB, h_matB, matB_size, hipMemcpyHostToDevice);
    hipMemcpy(d_matC, h_matC, matC_size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&sec, start, stop);
    printf(">> Host to GPU transfer time: %.3f\n", sec);

    sec = run_sgemm_naive(d_matA, d_matB, d_matC, d_matD, M, N, K, 0.7, 0.3);
    printf(">> Naive kernel execution time: %.3f\n", sec);
    hipMemcpy(h_matD, d_matD, matC_size, hipMemcpyDeviceToHost);

    sec = run_sgemm_coalesced(d_matA, d_matB, d_matC, d_matD, M, N, K, 0.7, 0.3);
    printf(">> Coalesced kernel execution time: %.3f\n", sec);
    hipMemcpy(h_matD, d_matD, matC_size, hipMemcpyDeviceToHost);

    sec = run_sgemm_shared_memory(d_matA, d_matB, d_matC, d_matD, M, N, K, 0.7, 0.3);
    printf(">> Shared memory kernel execution time: %.3f\n", sec);
    hipMemcpy(h_matD, d_matD, matC_size, hipMemcpyDeviceToHost);

    // hipEventRecord(start);
    // hipMemcpy(h_matD, d_matD, matC_size, hipMemcpyDeviceToHost);
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&sec, start, stop);
    // printf(">> GPU to host transfer time: %.3f\n", sec);

    // for(int i = 0; i < M * N; i++)
    //     printf("%f\n", h_matA[i]);
    // printf("----------------\n");
    // for(int i = 0; i < N * K; i++)
    //     printf("%f\n", h_matB[i]);
    // printf("----------------\n");
    // for(int i = 0; i < M * K; i++)
    //     printf("%f\n", h_matD[i]);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    free(h_matA);
    free(h_matB);
    free(h_matC);
    free(h_matD);

    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);
    hipFree(d_matD);

return 0;
}