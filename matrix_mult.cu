#include "hip/hip_runtime.h"
#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void matrix_mult(float *A, float*B, float *C, const int M, const int K, const int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y
    int col = blockIdx.x * blockDim.x + threadIdx.x

    if (row < M && col < N) {
        float sum = 0.0;
        for(int i = 0; i < K; i++)
            sum += A[row * K + i] * B[i * N + col];
        C[row * N + col] = sum;
    }
}

void random_init(float *matrix, int M, int N) {
    for(int i = 0; i < M * N; i++)
        matrix[i] = rand() % 10;
}

int main() {
    float *h_matA, *h_matB, *h_matC, *d_matA, *d_matB, *d_matC;
    int M = 2, N = 2, K = 2;
    int matAsize = sizeof(float) * M * K;
    int matBsize = sizeof(float) * K * N;
    int matCsize = sizeof(float) * M * N;

    h_matA = (float*)malloc(matAsize);
    h_matB = (float*)malloc(matBsize);
    h_matC = (float*)malloc(matCsize);

    random_init(h_matA, M, K);
    random_init(h_matB, K, N);

    hipMalloc((void**)&d_matA, matAsize);
    hipMalloc((void**)&d_matB, matBsize);
    hipMalloc((void**)&d_matC, matCsize);

    hipMemcpy(d_matA, h_matA, matAsize, hipMemcpyHostToDevice);
    hipMemcpy(d_matB, h_matB, matBsize, hipMemcpyHostToDevice);
    
    for(int i = 0; i < M * K; i++) 
        printf("\n%f", h_matA[i]);
    
    free(h_matA);
    free(h_matB);
    free(h_matC);

    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);
    return 0;
}