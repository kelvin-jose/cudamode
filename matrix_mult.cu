#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void matrix_mult(float *A, float*B, float *C, const int M, const int K, const int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0;
        for(int i = 0; i < K; i++)
            sum += A[row * K + i] * B[i * N + col];
        C[row * N + col] = sum;
    }
}

void random_init(float *matrix, int M, int N) {
    for(int i = 0; i < M * N; i++)
        matrix[i] = (float)rand() / RAND_MAX;
}

int main() {
    float *h_matA, *h_matB, *h_matC, *d_matA, *d_matB, *d_matC;
    int M = 256, N = 512, K = 128;
    int matA_size = sizeof(float) * M * K;
    int matB_size = sizeof(float) * K * N;
    int matC_size = sizeof(float) * M * N;

    h_matA = (float*)malloc(matA_size);
    h_matB = (float*)malloc(matB_size);
    h_matC = (float*)malloc(matC_size);

    random_init(h_matA, M, K);
    random_init(h_matB, K, N);

    hipMalloc((void**)&d_matA, matA_size);
    hipMalloc((void**)&d_matB, matB_size);
    hipMalloc((void**)&d_matC, matC_size);

    hipMemcpy(d_matA, h_matA, matA_size, hipMemcpyHostToDevice);
    hipMemcpy(d_matB, h_matB, matB_size, hipMemcpyHostToDevice);

    dim3 block_size(32, 32);
    dim3 grid_size((N + block_size.x - 1) / block_size.x, (M + block_size.y - 1) / block_size.y);

    matrix_mult<<<grid_size, block_size>>>(d_matA, d_matB, d_matC, M, K, N);

    hipMemcpy(h_matC, d_matC, matC_size, hipMemcpyDeviceToHost);
    
    for(int i = 0; i < M * N; i++) 
        printf("\n%f", h_matC[i]);

    free(h_matA);
    free(h_matB);
    free(h_matC);

    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);
    return 0;
}