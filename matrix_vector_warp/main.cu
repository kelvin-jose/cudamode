#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<warp_op.cuh>

void random_init(float *array, int M, int N) {
    for(int i = 0; i < M * N; i++)
        array[i] = (float)rand() / RAND_MAX;
}

int main() {
    srand(time(NULL));

    int M = 2, N = 2;

    float *h_matrix, *h_vector, *h_result;
    
    int matrix_size = M * N * sizeof(float);
    int vector_size = N * sizeof(float);

    h_matrix = (float*)malloc(matrix_size);
    h_vector = (float*)malloc(vector_size);
    h_result = (float*)malloc(vector_size);

    random_init(h_matrix, M, N);
    random_init(h_vector, N, 1);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    float sec = 0.0;
    float *d_matrix, *d_vector, *d_result;

    hipEventRecord(start);
    hipMalloc((void**)&d_matrix, matrix_size);
    hipMalloc((void**)&d_vector, vector_size);
    hipMalloc((void**)&d_result, vector_size);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&sec, start, stop);
    printf(">> GPU memory allocation time: %f\n", sec);

    hipEventRecord(start);
    hipMemcpy(d_matrix, h_matrix, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector, h_vector, vector_size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&sec, start, stop);
    printf(">> Host to GPU transfer time: %f\n", sec);

    sec = run_matrix_vector_mult(d_matrix, d_vector, d_result, M, N);
    printf(">> Kernel execution time: %f\n", sec);

    hipEventRecord(start);
    hipMemcpy(h_result, d_result, vector_size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&sec, start, stop);
    printf(">> GPU to host transfer time: %f\n", sec);

    for(int i = 0; i < M*N; i++)
        printf("%f\n", h_matrix[i]);
    printf("----------------\n");
    for(int i = 0; i < N; i++)
        printf("%f\n", h_vector[i]);
    printf("----------------\n");
    for(int i = 0; i < N; i++)
        printf("%f\n", h_result[i]);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    free(h_matrix);
    free(h_vector);
    free(h_result);

    hipFree(d_matrix);
    hipFree(d_vector);
    hipFree(d_result);

return 0;
}