#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include "utils.cuh"

__global__ void matrix_vector_mult(float *matrix, float *vector, float *result, int M, int N) {
    int block = blockIdx.x;
    if (block >= M)
        return;
    
    int thread = threadIdx.x;

    float sum = 0.0;

    for(int i = thread; i < N; i += blockDim.x) {
        sum += matrix[block * N + i] * vector[i];
    }

    sum = warp_reduce(sum);
    if (block == 0)
        result[block] = sum;
}

float run_matrix_vector_mult(float *matrix, float *vector, float *result, int M, int N) {

    dim3 block_size(WARP_SIZE);
    dim3 grid_size(M);

    hipEvent_t start, stop;
    float sec = 0.0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matrix_vector_mult<<<grid_size, block_size>>>(matrix, vector, result, M, N);
    hipEventRecord(stop);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&sec, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

return sec;
}