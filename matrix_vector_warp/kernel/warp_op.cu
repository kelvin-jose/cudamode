#include<hip/hip_runtime.h>

#define WARP_SIZE 32

__global__ void matrix_vector_mult(float *matrix, float *vector, float *result, int M, int N) {
    
}

float run_matrix_vector_mult(float *matrix, float *vector, float *result, int M, int N) {

    dim3 block_size(WARP_SIZE);
    dim3 grid_size(M);

    hipEvent_t start, stop;
    float sec = 0.0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matrix_vector_mult<<<grid_size, block_size>>>(matrix, vector, result, M, N);
    hipEventRecord(stop);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&sec, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

return sec;
}