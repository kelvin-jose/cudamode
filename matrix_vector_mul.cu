#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

int main() {
    srand(time(NULL));
    int N = 128;
    int *h_matrix, *h_vector, *d_matrix, *d_vector;
    size_t matrix_size = sizeof(int) * N * N;
    size_t vector_size = sizeof(int) * N;

    h_matrix = (int*)malloc(matrix_size);
    h_vector = (int*)malloc(vector_size);

    for(int i = 0; i < N * N; i++) 
        h_matrix[i] = rand() % 10;
    
    for(int i = 0;i < N; i++)
        h_vector[i] = rand() % 10;

    hipMalloc((void**)&d_matrix, matrix_size);
    hipMalloc((void**)&d_vector, vector_size);

    hipMemcpy(d_matrix, h_matrix, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector, h_vector, vector_size, hipMemcpyHostToDevice);


    printf("%d", h_matrix[10]);
    free(h_matrix);
    free(h_vector);
    hipFree(d_matrix);
    hipFree(d_vector);
    return 0;
}