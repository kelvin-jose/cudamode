#include<time.h>
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void matrix_vector_mult(int *matrix, int *vector, int *output, size_t N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < N) {
        int sum = 0;
        for(int j = 0; j < N; j++)
            sum += (matrix[i * N + j] * vector[j]);
        output[i] = sum;
    }
}

int main() {
    srand(time(NULL));
    int N = 256;
    int *h_matrix, *h_vector, *h_output, *d_matrix, *d_vector, *d_output;
    size_t matrix_size = sizeof(int) * N * N;
    size_t vector_size = sizeof(int) * N;

    h_matrix = (int*)malloc(matrix_size);
    h_vector = (int*)malloc(vector_size);
    h_output = (int*)malloc(vector_size);

    for(int i = 0; i < N * N; i++) 
        h_matrix[i] = rand() % 10;
    
    for(int i = 0; i < N; i++)
        h_vector[i] = rand() % 10;

    hipMalloc((void**)&d_matrix, matrix_size);
    hipMalloc((void**)&d_vector, vector_size);
    hipMalloc((void**)&d_output, vector_size);

    hipMemcpy(d_matrix, h_matrix, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector, h_vector, vector_size, hipMemcpyHostToDevice);

    dim3 block_size(64);
    dim3 grid_size((N + block_size.x - 1) / block_size.x);

    matrix_vector_mult<<<grid_size, block_size>>>(d_matrix, d_vector, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, vector_size, hipMemcpyDeviceToHost);

    for(int j = 0; j < N * N; j++) {
        if (j % N == 0)
            printf("\n");
        printf("%d ", h_matrix[j]);
    }
    
    for(int j = 0; j < N; j++)
        printf("\n%d", h_vector[j]);
    
    for(int i = 0; i < N; i++)
        printf("\n%d", h_output[i]);
    
    free(h_matrix);
    free(h_vector);
    free(h_output);
    hipFree(d_matrix);
    hipFree(d_vector);
    hipFree(d_output);
    return 0;
}