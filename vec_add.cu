#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void add_vectors(const float *a, const float *b, float *c, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
        c[idx] = a[idx] + b[idx];
}

int main() {
    
    // addition of two vectors in pure c
    int N = 100;
    float *a, *b, *c;

    float *d_a, *d_b, *d_c;

    // allocate memory
    a = (float *)malloc(N * sizeof(float));
    b = (float *)malloc(N * sizeof(float));
    c = (float *)malloc(N * sizeof(float));

    for(int i = 0; i<N; i++) {
        a[i] = 1.3 * i;
        b[i] = 2.5 * i;
        c[i] = a[i] + b[i];
    }

    for(int i=0; i<N; i++) 
        printf("%f\n", c[i]);
    
    hipMalloc((void **)&d_a, N * sizeof(float));
    hipMalloc((void **)&d_b, N * sizeof(float));
    hipMalloc((void **)&d_c, N * sizeof(float));

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, N * sizeof(float), hipMemcpyHostToDevice);


    return 0;
}